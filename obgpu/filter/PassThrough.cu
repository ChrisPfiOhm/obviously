#include "hip/hip_runtime.h"
#include "obgpu/filter/PassThrough.h"
#include "obgpu/PointCloud.h"

namespace obvious { namespace gpu { namespace filter {

__global__ void check_if_inside(obvious::gpu::PointXyz* input, bool* inside, float* limits)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    inside[i] = input[i].x >= limits[0] && input[i].x <= limits[1] && input[i].y >= limits[2] &&
        input[i].y <= limits[3] && input[i].z >= limits[4] && input[i].z <= limits[5];
}

void PassThrough::filter(obvious::gpu::PointCloud* cloud)
{
    hipMemcpy(_limits, &_xMin, 6 * sizeof(float), hipMemcpyHostToDevice);
    bool* inside;
    hipMalloc((void**)&inside, cloud->size() * sizeof(bool));

    check_if_inside<<<10, 1024>>>(reinterpret_cast<obvious::gpu::PointXyz*>(cloud->data()), inside, _limits);

    hipFree(inside);
}

} // end namespace filter

} // end namespace gpu

} // end namespace obvious
