#include "obgpu/PointCloud.h"
#include "obgpu/point-types.h"

#include <hip/hip_runtime_api.h>

namespace obvious { namespace gpu {

PointCloud::~PointCloud(void)
{
    if (_data)
        hipFree(_data);
}

void PointCloud::upload(const obvious::PointCloud<obvious::PointXyz>& cloud)
{
    if (_data && _type != XYZ)
    {
        hipFree(_data);
        _data = 0;
        _type = XYZ;
        _size = 0;
    }

    if (_size < cloud.size())
    {
        hipFree(_data);
        hipMalloc(&_data, cloud.size());
        _size = cloud.size();
    }
    else if (_size > cloud.size())
    {
        _size = cloud.size();
    }

    hipMemcpy(_data, cloud.points().data(), cloud.size() * sizeof(obvious::gpu::PointXyz), hipMemcpyHostToDevice);
}

void PointCloud::download(obvious::PointCloud<obvious::PointXyz>& cloud)
{
    if (cloud.size() != _size)
        cloud.resize(_size);

    hipMemcpy(cloud.points().data(), _data, cloud.size() * sizeof(obvious::gpu::PointXyz), hipMemcpyDeviceToHost);
}

} // end namespace gpu

} // end namespace obvious
